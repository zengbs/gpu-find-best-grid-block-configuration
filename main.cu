#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Size of array
#define N 1048570


unsigned long Time;

/* generate a random floating point number from min to max */
double randfrom(double min, double max)
{
    double range = (max - min);
    double div = RAND_MAX / range;
    return min + (rand() / div);
}

void Reset()
{
   Time = 0;
}

double GetValue()
{
   return Time*1.0e-6;
}

void Start()
{
   Time = 0;
   struct timeval tv;
   gettimeofday( &tv, NULL );
   Time   = tv.tv_sec*1000000 + tv.tv_usec;
}

void Stop()
{
   struct timeval tv;
   gettimeofday( &tv, NULL );
   Time   = tv.tv_sec*1000000 + tv.tv_usec - Time;
}

// Kernel
__global__ void kernel(double *a, double *b, double *c, long length)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    while(id < length){
       c[id] = a[id] + b[id] + sqrt(a[id]) + sqrt(b[id]);
       c[id] = sqrt(c[id]);
       id += gridDim.x*blockDim.x;
    }

}

// Main program
int main()
{
    srand (time ( NULL));

    // Number of bytes to allocate for N doubles
    size_t bytes = N*sizeof(double);

    // Allocate memory for arrays A, B, and C on host
    double *A = (double*)malloc(bytes);
    double *B = (double*)malloc(bytes);
    double *C = (double*)malloc(bytes);

    // Allocate memory for arrays d_A, d_B, and d_C on device
    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Fill host arrays A and B
    for(int i=0; i<N; i++)
    {
        A[i] = randfrom(+1.0, 16.0);
        B[i] = randfrom(+1.0, 4.0);
    }

    // Copy data from host arrays A and B to device arrays d_A and d_B
    hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

    // Set execution configuration parameters
    //      blockSize: number of CUDA threads per grid block
    //      gridSize: number of blocks in grid
    int blockSize = 1024;
    int gridSize = 1;

    // Warm up
    Start();
    kernel<<< gridSize, blockSize >>>(d_A, d_B, d_C, N);
    Stop();

    printf("# gridSize       blockSize        Time (sec)");

    // Launch kernel
    for ( gridSize = 1; gridSize<=1024; gridSize++ ){
       for ( blockSize = 1; blockSize<=1024; blockSize++ ){

          Start();
          kernel<<< gridSize, blockSize >>>(d_A, d_B, d_C, N);
          hipDeviceSynchronize();
          Stop();
          printf("%d, %d, %e\n", gridSize, blockSize, GetValue());

       }
       printf("\n");
    }


    // Free CPU memory
    free(A);
    free(B);
    free(C);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


    return 0;
}
